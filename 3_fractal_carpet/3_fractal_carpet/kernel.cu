#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ostream>
#include <iostream>
#include <array>
#include "hip/hip_runtime.h"
#include ""
#include "util.hpp"
#include "pfc_cuda_memory.h"
#include "pfc_cuda_exception.h"
#include "pfc_parallel.h"
#include "device.hpp"
#include <hip/device_functions.h>

CATTR_CONST pfc::bitmap::pixel_t gpu_color_mapping[RGB_COLOR_SIZE];

template <typename duration_t>
CATTR_HOST void display_results(std::string name, int tasks, duration_t duration) {
	auto const targetMillis = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

	std::cout << "name: " << name << " | block_size: {" << tasks << "," << tasks << "}" << " | millis: " << targetMillis << std::endl;
}

CATTR_KERNEL void fractal_kernel(pfc::complex<float> start,
	const int maxIterations,
	const int size,
	pfc::bitmap::pixel_t * result) {

	auto row{ __fmul_rn(blockIdx.x, blockDim.x) + threadIdx.x };
	auto col{ __fmul_rn(blockIdx.y,blockDim.y) + threadIdx.y };

	calculate_fractal_part(size, maxIterations, row, col, start, result, gpu_color_mapping);
}

CATTR_HOST void initialize_gpu() {
	int count{ 0 }; PFC_CUDA_CHECK(hipGetDeviceCount(&count));
	if (count > 0) {
		hipSetDevice(0);

		auto const deviceInfo{ pfc::cuda::get_device_info() };
		auto const deviceProps{ pfc::cuda::get_device_props() };

		std::cout << "-----------------------------------------------------" << std::endl;
		std::cout << "Device Metadata:" << std::endl;
		std::cout << "-----------------------------------------------------" << std::endl;
		std::cout << "Device: " << deviceProps.name << std::endl;
		std::cout << "Compute capability: " << deviceInfo.cc_major << "." << deviceInfo.cc_minor << std::endl;
		std::cout << "Arch: " << deviceInfo.uarch << std::endl;
		std::cout << "Cores: " << deviceInfo.cores_sm << std::endl;
		std::cout << "Global memory: " << deviceProps.totalGlobalMem / 1024 / 1024 << " MB" << std::endl;
		std::cout << "Shared memory per block: " << deviceProps.sharedMemPerBlock / 1024 << " KB" << std::endl;
		std::cout << "Shared memory per multiprocessor: " << deviceProps.sharedMemPerMultiprocessor / 1024 << " KB" << std::endl;
		std::cout << "Max threads per block: " << deviceInfo.max_threads_block << std::endl;
		std::cout << "-----------------------------------------------------" << std::endl;
		std::cout << std::endl;
	}
}

CATTR_HOST void inline execute_gpu_global_parallel_local_serial(const int pictureCount,
	const int size,
	const int maxIterations,
	dim3 block_size) {
	try {

		pfc::bitmap::pixel_t* device_pixels{ CUDA_MALLOC(pfc::bitmap::pixel_t, size*size) };
		//pfc::bitmap::pixel_t* device_rgb_map{ CUDA_MALLOC(pfc::bitmap::pixel_t, 16) };

		PFC_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gpu_color_mapping), &RGB_MAPPING, RGB_COLOR_SIZE));
		//CUDA_MEMCPY(gpu_color_mapping, RGB_MAPPING, RGB_COLOR_SIZE, hipMemcpyHostToDevice);

		for (int i = 0; i < pictureCount; ++i) {
			dim3 grid_size((size + block_size.x - 1) / block_size.x, (size + block_size.y - 1) / block_size.y);
			pfc::bitmap bitmap(size, size);
			fractal_kernel << < grid_size, block_size >> > (pfc::complex<float>(0, 0), maxIterations, size, device_pixels);
			PFC_CUDA_CHECK(hipGetLastError());
			PFC_CUDA_CHECK(hipDeviceSynchronize()); // synchronize with device, means wait for it
			PFC_CUDA_CHECK(hipGetLastError());
			PFC_CUDA_MEMCPY(bitmap.get_pixels(), device_pixels, size*size, hipMemcpyDeviceToHost);

			//bitmap.to_file(DIR_GPU_TEST + "fractal-gpu_" + std::to_string(block_size.x) + "_" + std::to_string(block_size.y) + "_" + std::to_string(i) + ".jpg");
		}

		CUDA_FREE(device_pixels);
	}
	catch (std::exception const &x) {
		std::cerr << x.what() << std::endl;
	}
}
CATTR_HOST void inline test_gpu_global_parallel_local_serial() {

	std::cout
		<< "#################################################################################" << std::endl
		<< "Start GPU tests 'GPLS'" << std::endl
		<< "#################################################################################" << std::endl;

	for each (unsigned int task_count in TASK_COUNTS)
	{

		auto duration_gpu = mpv_runtime::run_with_measure(1, [&]
		{
			execute_gpu_global_parallel_local_serial(PICTURE_COUNT, MAX_ITERATIONS, SIZE, dim3{ task_count ,task_count });
		});

		display_results("GPU-GPLS", task_count, duration_gpu);
	}

	std::cout
		<< "#################################################" << std::endl
		<< "Ended GPU tests 'GPLS'" << std::endl
		<< "#################################################" << std::endl;
}

int main()
{
	initialize_gpu();

	std::cout << std::endl;

	test_gpu_global_parallel_local_serial();
}


