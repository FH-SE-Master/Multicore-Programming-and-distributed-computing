#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <ostream>
#include <iostream>
#include <array>
#include "util.hpp"
#include "host_device.hpp"
#include "pfc_parallel.h"
#include "host.hpp"
#include "device.hpp"

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

int main()
{
	try
	{
		auto count{0};
		mpv_exception::check(hipGetDeviceCount(&count));
		if (count > 0)
		{
			hipSetDevice(0);

			auto const device_info{pfc::cuda::get_device_info()};
			auto const device_props{pfc::cuda::get_device_props()};

			std::cout << "Device            : " << device_props.name << std::endl;
			std::cout << "Compute capability: " << device_info.cc_major << "." << device_info.cc_minor << std::endl;
			std::cout << "Arch              : " << device_info.uarch << std::endl;
			std::cout << std::endl;

			// Test the host execution serial and parallel
			test_host();

			// Test the device execution
		}
	}
	catch (std::exception const& x)
	{
		std::cerr << x.what() << std::endl;
	}
}
