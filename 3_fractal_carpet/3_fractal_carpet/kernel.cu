#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <ostream>
#include <iostream>
#include <array>
#include "util.hpp"
#include "host_device.hpp"
#include "pfc_parallel.h"
#include "host.hpp"

const int HEIGHT = 1000;
const int WIDTH = HEIGHT;
const int MAX_ITERATIONS = 1000;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void fractal_kernel(int* c, const int* a, const int* b)
{
}

int main()
{

	try
	{
		int count{0};
		mpv_exception::check(hipGetDeviceCount(&count));
		if (count > 0)
		{
			hipSetDevice(0);

			auto const deviceInfo{pfc::cuda::get_device_info()};
			auto const deviceProps{pfc::cuda::get_device_props()};

			std::cout << "Device            : " << deviceProps.name << std::endl;
			std::cout << "Compute capability: " << deviceInfo.cc_major << "." << deviceInfo.cc_minor << std::endl;
			std::cout << "Arch              : " << deviceInfo.uarch << std::endl;
			std::cout << std::endl;

			execute_fractal_serial(HEIGHT, MAX_ITERATIONS);
			execute_fractal_parallel(4, HEIGHT, MAX_ITERATIONS);
			/*pfc::bitmap bitmap{WIDTH, HEIGHT};
			auto duration_thread_single = mpv_runtime::run_with_measure(1, [&]
		                                                            {
			                                                            calculate_fractal(HEIGHT, WIDTH, MAX_ITERATIONS, 0, 0, pfc::complex<float>(0,0), bitmap.get_pixels());
		                                                            });
			
			bitmap.to_file("fractal-0-0.jpg");

			std::cout << "CPU time (single thread): "
				<< std::chrono::duration_cast<std::chrono::milliseconds>(duration_thread_single).count() << " milliseconds" << std::
				endl << std::endl;
				*/
			
		}
	}
	catch (std::exception const& x)
	{
		std::cerr << x.what() << std::endl;
	}
}

